#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__ void copy(float *odata, float *idata, int N)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// printf("index: %i\n", index);

	if (index >= N * N)
		return ;
	for (int row = 0; row < N + 1; ++row)
		odata[row * N + index] = idata[row * N + index];
}

int main()
{

	int N = 2048;
	float *in, *out;

	hipMallocManaged(&in, N*N*sizeof(float));
	hipMallocManaged(&out, N*N*sizeof(float));

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			in[i * N + j] = j;

	hipMemPrefetchAsync(in, N*N*sizeof(float), 0, 0);

	int threadsPerBlock = 256;
	int blocks = (N * N + 255) / 256;

	copy<<<blocks, threadsPerBlock>>>(out, in, N);

	hipDeviceSynchronize();

	float maxError = 0.0f;

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			maxError = fmax(maxError, fabs(out[i * N + j] - j));

	std::cout << "max error: " << maxError << std::endl;

	hipFree(in);
	hipFree(out);

	return 0;
}
